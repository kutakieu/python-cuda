#include <raymarch.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include <vec3.h>
#include <camera.h>
#include <scene.h>
#include <distance_functions.h>
#include <thrust/device_vector.h>
using namespace std;

GPURenderer::GPURenderer(int *array_host_, int canvas_height_, int canvas_width_)
{
  cout << "here" << endl;
  array_host = array_host_;
  canvas_height = canvas_height_;
  canvas_width = canvas_width_;
  hipMallocManaged((void **)&fb, sizeof(vec3) * canvas_height * canvas_width);
  // scene = make_scene();
  Sphere s = Sphere(1.0);
  // scene.add_object(s);
  thrust::device_vector<Sphere> objects;
  objects.push_back(s);
  int n_objects = objects.size();
  camera cam = camera(vec3(0, 1, 0), vec3(0, 0, -1), vec3(0, 0, 2), 90);
  // scene = Scene(cam, thrust::raw_pointer_cast(&objects[0]), n_objects);
  scene = Scene(cam, s, n_objects);
  cout << "n_objects: " << n_objects << endl;
}

void GPURenderer::render()
{
  int tx = 8;
  int ty = 8;
  dim3 blocks(canvas_width / tx + 1, canvas_height / ty + 1);
  dim3 threads(tx, ty);
  // cam = camera(vec3(0, 1, 0), vec3(0, 0, -1), vec3(0, 0, 2), 90);
  // scene = Scene();
  // Sphere s = Sphere(1.0);
  // scene.add_object(s);
  // scene.copy_to_device();
  kernel_ray_marching<<<blocks, threads>>>(fb, canvas_height, canvas_width, scene);
  hipError_t err = hipGetLastError();
  assert(err == 0);
  hipDeviceSynchronize();
  err = hipGetLastError();
  assert(err == 0);
  fb2img();
  hipFree(fb);
}

// Scene GPURenderer::make_scene()
// {
//   Sphere s = Sphere(1.0);
//   scene.add_object(s);
//   // scene.copy_to_device();
//   return scene;
// }

void GPURenderer::fb2img()
{
  for (int j = canvas_height - 1; j >= 0; j--)
  {
    for (int i = 0; i < canvas_width; i++)
    {
      size_t pixel_index = j * canvas_width + i;
      int ir = int(255.99 * fb[pixel_index].r());
      int ig = int(255.99 * fb[pixel_index].g());
      int ib = int(255.99 * fb[pixel_index].b());
      array_host[pixel_index * 3] = ir;
      array_host[pixel_index * 3 + 1] = ig;
      array_host[pixel_index * 3 + 2] = ib;
    }
  }
}

GPURenderer::~GPURenderer()
{
  hipFree(array_device);
}
