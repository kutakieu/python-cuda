#include "hip/hip_runtime.h"
#include <stdio.h>
#include "vec3.h"

void __global__ kernel_add_one(int *a, int length)
{
    int gid = threadIdx.x + blockDim.x * blockIdx.x;

    while (gid < length)
    {
        a[gid] += 1;
        gid += blockDim.x * gridDim.x;
    }
}

__global__ void kernel_render(vec3 *fb, int max_x, int max_y)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y))
        return;
    int pixel_index = j * max_x + i;
    fb[pixel_index] = vec3(float(i) / max_x, float(j) / max_y, 0.2f);
}
