/*
This is the central piece of code. This file implements a class
(interface in RayMarcher.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU

This class will get translated into python via swig
*/

#include <kernel.cu>
#include <ray_marcher.hh>
#include <assert.h>
#include <iostream>
#include <vec3.h>
using namespace std;

RayMarcher::RayMarcher(int *array_host_, int canvas_height_, int canvas_width_)
{
  array_host = array_host_;
  canvas_height = canvas_height_;
  canvas_width = canvas_width_;
  hipMallocManaged((void **)&fb, sizeof(vec3) * canvas_height * canvas_width);
}

void RayMarcher::render()
{
  int tx = 8;
  int ty = 8;
  dim3 blocks(canvas_width / tx + 1, canvas_height / ty + 1);
  dim3 threads(tx, ty);
  kernel_render<<<blocks, threads>>>(fb, canvas_height, canvas_width);
  hipError_t err = hipGetLastError();
  assert(err == 0);
  hipDeviceSynchronize();
  err = hipGetLastError();
  assert(err == 0);

  for (int j = canvas_height - 1; j >= 0; j--)
  {
    for (int i = 0; i < canvas_width; i++)
    {
      size_t pixel_index = j * canvas_width + i;
      int ir = int(255.99 * fb[pixel_index].r());
      int ig = int(255.99 * fb[pixel_index].g());
      int ib = int(255.99 * fb[pixel_index].b());
      array_host[pixel_index * 3] = ir;
      array_host[pixel_index * 3 + 1] = ig;
      array_host[pixel_index * 3 + 2] = ib;
    }
  }
  hipFree(fb);
}

void RayMarcher::increment()
{
  kernel_add_one<<<64, 64>>>(array_device, length);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

void RayMarcher::retreive()
{
  int size = length * sizeof(int);
  hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  if (err != 0)
  {
    cout << err << endl;
    assert(0);
  }
}

void RayMarcher::retreive_to(int *array_host_, int length_)
{
  assert(length == length_);
  int size = length * sizeof(int);
  hipMemcpy(array_host_, array_device, size, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

RayMarcher::~RayMarcher()
{
  hipFree(array_device);
}
